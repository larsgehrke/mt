#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include <config.h>

namespace {
template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
  const auto s = sigmoid(z);
  return (1.0 - s) * s;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_tanh(scalar_t z) {
  const auto t = tanh(z);
  return 1 - (t * t);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t elu(scalar_t z, scalar_t alpha = 1.0) {
  return fmaxf(0.0, z) + fminf(0.0, alpha * (exp(z) - 1.0));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_elu(scalar_t z, scalar_t alpha = 1.0) {
  const auto e = exp(z);
  const auto d_relu = z < 0.0 ? 0.0 : 1.0;
  return d_relu + (((alpha * (e - 1.0)) < 0.0) ? (alpha * e) : 0.0);
}

template <typename scalar_t>
__global__ void distana_cuda_forward_kernel(
    /*const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> gates,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> old_cell,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_h,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_cell,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input_gate,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output_gate,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> candidate_cell*/
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input) {
  
    // PKs ?
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    // Batch ?
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    input_gate[y][x][0][0] = -7;
}

template <typename scalar_t>
__global__ void distana_cuda_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_old_cell,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> d_gates,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> grad_h,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> grad_cell,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_cell,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input_gate,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output_gate,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> candidate_cell,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> gate_weights) {
  //batch index
  const int n = blockIdx.y;
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < d_gates.size(2)){
    const auto d_output_gate = tanh(new_cell[n][c]) * grad_h[n][c];
    const auto d_tanh_new_cell = output_gate[n][c] * grad_h[n][c];
    const auto d_new_cell =
        d_tanh(new_cell[n][c]) * d_tanh_new_cell + grad_cell[n][c];


    d_old_cell[n][c] = d_new_cell;
    const auto d_candidate_cell = input_gate[n][c] * d_new_cell;
    const auto d_input_gate = candidate_cell[n][c] * d_new_cell;

    d_gates[n][0][c] =
        d_input_gate * d_sigmoid(gate_weights[n][0][c]);
    d_gates[n][1][c] =
        d_output_gate * d_sigmoid(gate_weights[n][1][c]);
    d_gates[n][2][c] =
        d_candidate_cell * d_elu(gate_weights[n][2][c]);
  }
}
} // namespace

std::vector<torch::Tensor> distana_cuda_forward(
    torch::Tensor input,
    torch::Tensor pre_weights,
    torch::Tensor lstm_weights,
    torch::Tensor post_weights,
    torch::Tensor old_h,
    torch::Tensor old_cell) {
  // Concatinates the tensors in the given dimensionality
  //auto X = torch::cat({old_h, input}, /*dim=*/1);

  // Performs a matrix multiplication of the matrices X and weights.transpose. 
  // The matrix bias is added to the final result.
  //auto gate_weights = torch::addmm(bias, X, pre_weights.transpose(0, 1));

  //const auto batch_size = input.size(0);
  //const auto state_size = old_cell.size(1);

  //auto gates = gate_weights.reshape({batch_size, 3, state_size});
  auto new_h = torch::zeros_like(old_cell);
  auto new_cell = torch::zeros_like(old_cell);
  auto input_gate = torch::zeros_like(old_cell);
  auto output_gate = torch::zeros_like(old_cell);
  auto candidate_cell = torch::zeros_like(old_cell);
  auto X = torch::zeros_like(input);
  auto new_pre_weights = torch::zeros_like(pre_weights);
  auto new_lstm_weights = torch::zeros_like(lstm_weights);
  auto new_post_weights = torch::zeros_like(post_weights);


  const int threads = BATCH_SIZE;
  const dim3 blocks(PK_ROWS, PK_COLS);

  AT_DISPATCH_FLOATING_TYPES(gates.type(), "distana_forward_cuda", ([&] {
    distana_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
      input.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>()
       /* gates.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        old_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_h.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        input_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        output_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        candidate_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>()*/
        );
  }));

  return {new_h, new_cell, input_gate, output_gate, candidate_cell, X, new_pre_weights, lstm_weights, post_weights};
}

std::vector<torch::Tensor> distana_cuda_backward(
    torch::Tensor grad_h,
    torch::Tensor grad_cell,
    torch::Tensor new_cell,
    torch::Tensor input_gate,
    torch::Tensor output_gate,
    torch::Tensor candidate_cell,
    torch::Tensor X,
    torch::Tensor gates,
    torch::Tensor weights) {
  auto d_old_cell = torch::zeros_like(new_cell);
  auto d_gates = torch::zeros_like(gates);

  const auto batch_size = new_cell.size(0);
  const auto state_size = new_cell.size(1);

  const int threads = 1024;
  const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(X.type(), "distana_forward_cuda", ([&] {
    distana_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        d_old_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        d_gates.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        grad_h.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        grad_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        input_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        output_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        candidate_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        gates.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
  }));

  auto d_gate_weights = d_gates.flatten(1, 2);
  auto d_weights = d_gate_weights.t().mm(X);
  auto d_bias = d_gate_weights.sum(/*dim=*/0, /*keepdim=*/true);

  auto d_X = d_gate_weights.mm(weights);
  auto d_old_h = d_X.slice(/*dim=*/1, 0, state_size);
  auto d_input = d_X.slice(/*dim=*/1, state_size);

  return {d_old_h, d_input, d_weights, d_bias, d_old_cell, d_gates};
}
